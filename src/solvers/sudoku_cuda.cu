#include "hip/hip_runtime.h"
// #define _GNU_SOURCE

#include "bitfield.cuh"
#include "wfc.cuh"
#include "utils.cuh"

#include <cstdio>
#include <cstdlib>


// #if !defined(WFC_CUDA)
// #error "WDC_CUDA should be defined..."
// #endif
__global__ void
solve_cuda_device(wfc_blocks_ptr blocks)
{
    uint64_t iteration  = 0;
    const uint64_t seed = blocks->seed;
    // struct {
    //     uint32_t gy, x, y, _1;
    //     uint64_t state;
    // } row_changes[blocks->grid_side];

    // grd_print(NULL, blocks);
    // getchar();

    bool success = false;

    jusqua_la_retraite {

        // grd_print(NULL, blocks);

        vec4 loc = grd_min_entropy(blocks);
        // printf(" choose loc :   [%d, %d] : [%d, %d] = %lu\n", 
        // loc.gy, loc.gx, loc.y, loc.x, *state );
      
        if( loc.x == UINT8_MAX ){
            success = true;
            // printf("success\n");
            break;
        }

        uint64_t * state = blk_at(blocks, loc.gx, loc.gy,
                                      loc.x, loc.y);
 

        if( state == 0){
            // printf("state = 0\n");
            break;
        }

        uint64_t collapsed_state = entropy_collapse_state(
            *state, loc.gx, loc.gy, loc.x, loc.y,
            blocks->seed, iteration);
        *state = collapsed_state;

        bool error = grd_propagate_all(blocks, loc.gx,
                            loc.gy, loc.x, loc.y, collapsed_state);
        

        if( error ){
            // printf("error\n");
            break;
        }

        iteration += 1;
    }
    
    if(success){
	grd_print(NULL, blocks);
    }
    // getchar();
    blocks->solved = success; 
    return ;
}

bool
solve_cuda(wfc_blocks_ptr blocks)
{

    printf("solver: addr block : %p\n", blocks);

    checkCudaErrors(hipGetLastError());
    solve_cuda_device<<<1,1>>>(blocks);

    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    wfc_blocks_ptr b = (wfc_blocks*) malloc(sizeof(wfc_blocks));
    hipMemcpy(b, blocks, sizeof(wfc_blocks), hipMemcpyDeviceToHost);
    printf("hello world from Host, seccess ? : %u \n", b->solved);
    // grd_print(NULL, b);
    bool succes = b->solved;
    free(b);
    return succes;
}
